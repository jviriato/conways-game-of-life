#include "hip/hip_runtime.h"
#include "bmp.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>


// Número de threads por bloco
#define BLOCK_SIZE 16

/*
 * Tempo (wallclock) em microssegundos
 */
 long wtime() {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1000000 + t.tv_usec;
  }


__global__ void ghostRows(int grid_size, int *grid) {
  int id = blockDim.x * blockIdx.x + threadIdx.x + 1;

  if (id <= grid_size) {
    // copia a primeira coluna para o fim
    grid[(grid_size + 2) * (grid_size + 1) + id] = grid[(grid_size + 2) + id];
    // copia a última coluna para o topo
    grid[id] = grid[(grid_size + 2) * grid_size + id];
  }
}

__global__ void ghostCols(int grid_size, int *grid) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id <= grid_size + 1) {
    // copia a primeira coluna para a direita
    grid[id * (grid_size + 2) + grid_size + 1] = grid[id * (grid_size + 2) + 1];
    // copia a última coluna para a esquerda
    grid[id * (grid_size + 2)] = grid[id * (grid_size + 2) + grid_size];
  }
}

__global__ void crossRules(int grid_size, int *grid, int *newGrid) {
  int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
  int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int id = iy * (grid_size + 2) + ix;

  int numNeighbors;

  if (iy <= grid_size && ix <= grid_size) {
    numNeighbors = grid[id + (grid_size + 2)] + grid[id - (grid_size + 2)]   
                   + grid[id + 1] + grid[id - 1]                 
                   + grid[id + (grid_size + 3)] + grid[id - (grid_size + 3)]
                   + grid[id - (grid_size + 1)] + grid[id + (grid_size + 1)];

    int cell = grid[id];
    if (cell == 1 && numNeighbors < 2)
      newGrid[id] = 0;
    else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
      newGrid[id] = 1;
    else if (cell == 1 && numNeighbors > 3)
      newGrid[id] = 0;
    else if (cell == 0 && numNeighbors == 3)
      newGrid[id] = 1;
    else
      newGrid[id] = cell;
  }
}

int main(int argc, char *argv[]) {
  int generations, grid_size;
  if (argc == 3) {
    generations = atoi(argv[1]);
    grid_size = atoi(argv[2]);
  } else {
    printf("args: generations and grid_size");
    return -1;
  }
  int *h_grid;    
  int *d_grid;    
  int *d_newGrid; 
  int *d_tmpGrid; 

  size_t bytes =
      sizeof(int) * (grid_size + 2) *
      (grid_size + 2);
  h_grid = (int *)malloc(bytes);

  hipMalloc(&d_grid, bytes);
  hipMalloc(&d_newGrid, bytes);
  long start_time = wtime();

  srand((unsigned long)time(NULL));
  for (int i = 1; i <= grid_size; i++) {
    for (int j = 1; j <= grid_size; j++) {
      h_grid[i * (grid_size + 2) + j] = rand() % 2;
    }
  }
  hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);

  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
  int linGrid = (int)ceil(grid_size / (float)BLOCK_SIZE);
  dim3 gridSize(linGrid, linGrid, 1);

  dim3 cpyBlockSize(BLOCK_SIZE, 1, 1);
  dim3 cpyGridRowsGridSize((int)ceil(grid_size / (float)cpyBlockSize.x), 1, 1);
  dim3 cpyGridColsGridSize((int)ceil((grid_size + 2) / (float)cpyBlockSize.x), 1, 1);

  for (int i  = 0; i < generations; i++) {
    ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(grid_size, d_grid);
    ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(grid_size, d_grid);
    crossRules<<<gridSize, blockSize>>>(grid_size, d_grid, d_newGrid);
    d_tmpGrid = d_grid;
    d_grid = d_newGrid;
    d_newGrid = d_tmpGrid;
  }

  hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
  long end_time = wtime();
  printf("%ld usec\n", (long)(end_time - start_time));

  int total = 0;
  for (int i = 1; i <= grid_size; i++) {
    for (int j = 1; j <= grid_size; j++) {
      total += h_grid[i * (grid_size + 2) + j];
    }
  }
  printf("Total Alive: %d\n", total);


  hipFree(d_grid);
  hipFree(d_newGrid);
  free(h_grid);

  return 0;
}